
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <chrono>
#include <iostream>

#define WIDTH 64  // Ends up being squared for the proper calculation (e.g. SIZE 4 means 4x4 matrices)
#define TILE_WIDTH 16
#define RUNS 100

// function to calculate the scan on GPU
__global__ void matmul(float *M, float *N, float *P, int width){
  int col = blockIdx.x*blockDim.x+threadIdx.x; // cols are .x in grid setup
  int row = blockIdx.y*blockDim.y+threadIdx.y; // rows are .y in grid setup

  if (row < width && col < width) { // inside range
    float sum = 0;
    for (int k=0; k < width; k++) {
      float a = M[ row*width + k ];
      float b = N[ k*width + col ];
      sum += a*b;
    }
    P[ row*width + col ] = sum;
  }
}

int main() {
  int SIZE = WIDTH*WIDTH; // compatibility with ported code from scan. since matrices are being initialized as 1D arrays, need a single size var. i'd prefer 2D. just sayin'
  std::cout << "\n" << SIZE << ","; // record the size of the run for data collection
  
  // allocate input and output arrays
  float *M, *d_M, *N, *d_N, *P, *d_P;
  M = (float*) malloc(SIZE*sizeof(float)); N = (float*) malloc(SIZE*sizeof(float)); P = (float*) malloc(SIZE*sizeof(float));
  hipMalloc(&d_M, SIZE*sizeof(float)); hipMalloc(&d_N, SIZE*sizeof(float)); hipMalloc(&d_P, SIZE*sizeof(float));

  // initialize inputs
  for (int j = 0; j < SIZE; j++) {
    M[j] = 1;
    N[j] = 1;
  }
  hipMemcpy(d_M, M, SIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_N, N, SIZE*sizeof(float), hipMemcpyHostToDevice);

  // initialize grid for indexing in matmul
  float* temp; temp = (float*) malloc(sizeof(float)); // don't ask why i do the things i do in code.
  temp[0] = ceil( (1.*WIDTH)/TILE_WIDTH ); // normal dereferencing for a pointer that is not an array.
  dim3 dimGrid(temp[0], temp[0], 1); // this line is :sparkle: magic :sparkle:
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  free(temp); // ...annndddd she's gone. out of sight, out of mind. just like she said. on the tag.

  // time it a bunch of times
  for (int i = 0; i < RUNS; i++) {
    
    const auto start{std::chrono::steady_clock::now()};
    matmul<<< dimGrid, dimBlock >>>(d_M, d_N, d_P, WIDTH);
    hipDeviceSynchronize(); // patience, girls
    const auto end{std::chrono::steady_clock::now()};
    const std::chrono::duration<double> elapsed{end - start};
    std::cout << elapsed.count() << "\n";

    // this isn't really part of the operation of matmul so she doesn't get timed. i know. she's missing out. it's ok though, she's not competitive.
    hipMemcpy(P, d_P, SIZE*sizeof(float), hipMemcpyDeviceToHost);
  }

  // check results
  for (int i = 0; i < SIZE; i++) {
    if (P[i] != WIDTH) { std::cerr << "IDX: " << i << "   OUT: " << P[i] << "   EXP: " << WIDTH << "\n"; }
  }

  // free mem
  free(M);
  hipFree(d_M);
  free(N);
  hipFree(d_N);
  free(P);
  hipFree(d_P);

  return 0;
}
